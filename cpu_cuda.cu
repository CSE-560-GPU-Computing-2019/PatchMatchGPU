#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
#include <stdio.h>
#include <time.h>

#define imgchannels 3
#define maskCols 5
#define maskRows 5
// #define THRESHOLD 100 // In percentage
#define MAXLEN 1024 // Max length of image paths
#define IMGSIZE 128
#define THRESHOLD_GPU 200
#define BLOCKSIZESQ 32

int THRESHOLD = 0;

// To access grayscale value at (i, j) or (x, y) do x + y * img_width

// To access rgb values of colored image, on the returned value index 0 is R, index 1 if G and 2 is B
__host__ __device__
unsigned char *getRGBOffset(int i, int j, unsigned char *c_image, int img_height, int img_width) {
    return c_image + (i + img_height * j) * 3;
}

// To access rgb values of colored image, on the returned value index 0 is R, index 1 if G and 2 is B
unsigned char *getIMGOffset(int i, int j, unsigned char *c_image, int img_height, int img_width) {
    return c_image + (i + img_height * j);
}


// Send offset of image to the beginning or top-left of the starting of the grid.
__host__ __device__
int compareGridsEachPixel(const unsigned char *c_as_g_image, const unsigned char *g_image, const unsigned char *c_as_g_image_BASE, const unsigned char *g_image_BASE, int gridSizeX, int gridSizeY, int c_width, int c_height, int g_width, int g_height) {
    int sum_c_as_g = 0;
    int sum_g = 0;
    int absDiff = 0;

    for (int row = -2; row < gridSizeY/2; ++row) {
        for (int col = -2; col < gridSizeX/2; ++col) {
            if (c_as_g_image + col + row * c_width < c_as_g_image_BASE)
                continue;
            if (g_image + col + row * g_width < g_image_BASE)
                continue;
            if (c_as_g_image + col + row * c_width > c_as_g_image_BASE + c_width * c_height)
                continue;
            if (g_image + col + row * g_width > g_image_BASE + g_width * g_height)
                continue;
            sum_c_as_g = c_as_g_image[col + row * c_width];
            sum_g = g_image[col + row * g_width];
            absDiff += abs(sum_c_as_g - sum_g);
        }
        // printf("Value of abs at row %d is %d\n", row, absDiff);
    }
    // printf("Value of absDiff is %d\n", absDiff);
    return absDiff;
}



/**
 * Provide correct offsets of final Image this code assumes that finalImage point to the grid where the color it to be copied to.
 * Same goes for c_image COLORS PIXEL INSTEAD OF GRID
 */
__host__ __device__
void colorImagePatchEachPixel(unsigned char *finalImage, unsigned char *c_image, int gridSizeX, int gridSizeY, int c_width, int c_height, int g_width, int g_height) {
    unsigned char *c_image_pixel;
    unsigned char *finalImage_pixel;

    // if (finalImage + 2 * dataSizeX + 2 > finalImage)
    //     return;
    // if (c_image + 2 * dataSizeX + 2 > strlen(c_image))
    //     return;

    c_image_pixel = getRGBOffset(0, 0, c_image, c_height, c_width);
    finalImage_pixel = getRGBOffset(0, 0, finalImage, g_height, g_width);
    // finalImage[col + row * gridSizeX] = ;
    finalImage_pixel[0] = c_image_pixel[0]; // Copy R
    finalImage_pixel[1] = c_image_pixel[1]; // Copy G
    finalImage_pixel[2] = c_image_pixel[2]; // Copy B
    // for (int row = 0; row < gridSizeY; ++row) {
    //     for (int col = 0; col < gridSizeX; ++col) {
    //         // printf("Row in colorImagePatch is: %d, col is: %d\n", row, col);
    //         c_image_pixel = getRGBOffset(col, row, c_image, dataSizeY, dataSizeX);
    //         finalImage_pixel = getRGBOffset(col, row, finalImage, dataSizeY, dataSizeX);
    //         // finalImage[col + row * gridSizeX] = ;
    //         finalImage_pixel[0] = c_image_pixel[0]; // Copy R
    //         finalImage_pixel[1] = c_image_pixel[1]; // Copy G
    //         finalImage_pixel[2] = c_image_pixel[2]; // Copy B
    //     }
    // }
    // printf("HEREBOID\n");
}

__global__
void gpuPathMatchEachPixel(unsigned char *c_image, const unsigned char *c_as_g_image, const unsigned char *g_image, unsigned char *finalImage, int gridSizeX, int gridSizeY, int c_width, int c_height, int g_width, int g_height)
{
    __shared__ int absDiffGrid[BLOCKSIZESQ][BLOCKSIZESQ];
    int c_as_g_index_row = 0;
    int c_as_g_index_col = 0;
    int g_index_row = 0;
    int g_index_col = 0;
    int absDiff = 0;

    g_index_row = threadIdx.y + blockIdx.y * blockDim.y;
    g_index_col = threadIdx.x + blockIdx.x * blockDim.x;

    // printf("OPA\n");

    for (int row = 0; row < c_height; ++row) { // Iterate over c_as_g_image
        // c_as_g_index_row = row * gridSizeX;
        c_as_g_index_row = row;
        for (int col = 0; col < c_width; ++col) { // Iterate over c_as_g_image
            // c_as_g_index_col = col * gridSizeY;
            c_as_g_index_col = col;

            absDiff = compareGridsEachPixel(c_as_g_image + c_as_g_index_col + (c_as_g_index_row * c_width), 
                                                    g_image + g_index_col + (g_index_row * g_width), 
                                                    c_as_g_image,
                                                    g_image,
                                                    gridSizeX, 
                                                    gridSizeY,
                                                    c_width, c_height, g_width, g_height);
            // printf("OPA\n");
            
            if (absDiff < THRESHOLD_GPU) {
                // if (finalImage[g_index_col + g_index_row * gridSizeX] == '\0') {
                // printf("g_index_row %d, g_index_col %d\n", g_index_row, g_index_col);
                if (absDiffGrid[threadIdx.y][threadIdx.x] == 0) {
                //     // colorImagePatch(finalImage,
                //     //                 c_image,
                //     //                 gridSizeX, 
                //     //                 gridSizeY,
                //     //                 dataSizeX,
                //     //                 dataSizeY);
                //     // colorImagePatch(finalImage + g_index_col + (g_index_row * dataSizeX),
                //     //                 c_image + c_as_g_index_col + (c_as_g_index_row * dataSizeX),
                //     //                 gridSizeX,
                //     //                 gridSizeY,
                //     //                 dataSizeX,
                //     //                 dataSizeY);
                    colorImagePatchEachPixel(getRGBOffset(g_index_col, g_index_row, finalImage, g_width, g_height),
                                    getRGBOffset(c_as_g_index_col, c_as_g_index_row, c_image, c_width, c_height),
                                    gridSizeX,
                                    gridSizeY,
                                    c_width, c_height, g_width, g_height);
                //     // absDiffGrid[g_index_row][g_index_col] = absDiff;
                    absDiffGrid[threadIdx.y][threadIdx.x] = absDiff; // g_index_row and g_index_col because the above commented line was going out of scope because absDiff is reduced size grid (check at top)
                //     // printf("BOIBOI\n");
                } 
                else if (absDiff < absDiffGrid[threadIdx.y][threadIdx.x]){ // If new absDiff < previousAbsDiff then update
                    colorImagePatchEachPixel(getRGBOffset(g_index_col, g_index_row, finalImage, g_width, g_height),
                                    getRGBOffset(c_as_g_index_col, c_as_g_index_row, c_image, c_width, c_height),
                                    gridSizeX,
                                    gridSizeY,
                                    c_width, c_height, g_width, g_height);
                    absDiffGrid[threadIdx.y][threadIdx.x] = absDiff;
                }
            }

        }

    }
}


void generatePathNames(char *sizeOfAllImage, char *grayscaleInputName, char *coloredImageName, 
                       char *coloredAsGrayscaleImageName, char *grayscaleImagePath,
                       char *coloredImagePath, char *coloredAsGrayscaleImagePath,
                       char * outputImagePath,
                       char * outputImagePathGPU) {
    char folderName[] = "Images";
    char input_to_be_colored[] = "input_to_be_colored";
    char input_color_name[] = "input_color";
    char input_grayscale_name[] = "input_grayscale";
    char ch;
    snprintf(grayscaleImagePath, MAXLEN, "%s/%s/%s/%s", folderName, sizeOfAllImage, input_to_be_colored, grayscaleInputName);
    snprintf(coloredImagePath, MAXLEN, "%s/%s/%s/%s", folderName, sizeOfAllImage, input_color_name, coloredImageName);
    snprintf(coloredAsGrayscaleImagePath, MAXLEN, "%s/%s/%s/%s", folderName, sizeOfAllImage, input_grayscale_name, coloredAsGrayscaleImageName);
    snprintf(outputImagePath, MAXLEN, "%s/%s/%s/%s", folderName, sizeOfAllImage, "output", grayscaleInputName);
    snprintf(outputImagePathGPU, MAXLEN, "%s/%s/%s/", folderName, sizeOfAllImage, "output");
    // snprintf(outputImagePathGPU, MAXLEN, "%s/%s/", folderName, sizeOfAllImage);
    strncat(outputImagePathGPU, grayscaleInputName, strrchr(grayscaleInputName, '.') - grayscaleInputName);
    // strncat(outputImagePath, grayscaleInputName, strrchr(grayscaleInputName, '.') - grayscaleInputName);
    // strcat(outputImagePath, "_colored.jpg");
    strcat(outputImagePathGPU, "_GPU.jpg");

    // printf("Input . start at %s\n", strrchr(grayscaleInputName, '.'));


    printf("Path generated are: \n");
    printf("grayscaleImagePath: %s\n", grayscaleImagePath);
    printf("coloredImagePath: %s\n", coloredImagePath);
    printf("coloredAsGrayscaleImagePath: %s\n", coloredAsGrayscaleImagePath);
    // printf("outputImagePath: %s\n", outputImagePath);
    printf("outputImagePathGPU: %s\n", outputImagePathGPU);
    // printf("Press any key to continue\n");
    // scanf("%c",&ch);
}

void copyGrayscaleToFinal(unsigned char * finalImage, const unsigned char *g_image, int dataSizeX, int dataSizeY) {
    for (int i = 0; i < dataSizeX; ++i) {
        for (int j = 0; j < dataSizeY; ++j) {
            // if (finalImage[i + j * dataSizeX] != '\0')
                // continue;
            finalImage[(i + dataSizeX * j) * 3] = g_image[i + j * dataSizeX];
            finalImage[1 + (i + dataSizeX * j) * 3] = g_image[i + j * dataSizeX];
            finalImage[2 + (i + dataSizeX * j) * 3] = g_image[i + j * dataSizeX];
        }
    }
}

void convertToGrayscale(unsigned char *finalImage, const unsigned char *g_image, int dataSizeX, int dataSizeY)
{
    for (int i = 0; i < dataSizeX; ++i)
    {
        for (int j = 0; j < dataSizeY; ++j)
        {
            // printf("%d, %d, %d\n", g_image[(i + dataSizeX * j) * 3], g_image[1 + (i + dataSizeX * j) * 3], g_image[2 + (i + dataSizeX * j) * 3]);
            int avg = (g_image[(i + dataSizeX * j) * 3] + g_image[1 + (i + dataSizeX * j) * 3] + g_image[2 + (i + dataSizeX * j) * 3]) / 3;
            // printf("Avg: %d\n", avg);
            finalImage[i + dataSizeX * j] = avg;
            finalImage[1 + i + dataSizeX * j] = avg;
            finalImage[2 + i + dataSizeX * j] = avg;
        }
    }
}

// g -> Grayscale
// c -> Color
int main(int argc, char *argv[]){
    if (argc != 2) {
        printf("Usage: ./cpu <THRESHOLD>");
        return 1;
    }
    THRESHOLD = atoi(argv[1]);
    char sizeOfAllImage[] = "128"; // Must be a square image and all must be of the same size
    char grayscaleInputName[] = "1.jpg";                    // Image to be colored
    char coloredImageName[] = "1.jpg";                     // Image from which color will be taken
    char coloredAsGrayscaleImageName[] = "1.jpg";          // The coloredImage changed to grayscale.
    // char coloredAsGrayscaleImageName[] = "converted_color_";

    char grayscaleImagePath[MAXLEN] = {};
    char coloredImagePath[MAXLEN] = {};
    char coloredAsGrayscaleImagePath[MAXLEN] = {};
    char outputImagePath[MAXLEN] = {};
    char outputImagePathGPU[MAXLEN] = {};

    // Cuda variables
    hipEvent_t start, stop;
    float elapsedTime;
    unsigned char *d_c_image;
    unsigned char *d_c_as_g_image;
    unsigned char *d_g_image;
    unsigned char *d_finalImage;
    

    int c_width, c_height, c_bpp; // For reading color image
    int c_as_g_width, c_as_g_height, c_as_g_bpp; // For reading grayscale of color iamge. Don't have a way to convert to grayscale in the code.
    int g_width, g_height, g_bpp; // Image to be colored
    unsigned char *finalImage, *c_as_g_image_load; // To be written
    unsigned char *finalImageByGPU;


    generatePathNames(sizeOfAllImage, grayscaleInputName, coloredImageName, coloredAsGrayscaleImageName, grayscaleImagePath, coloredImagePath, coloredAsGrayscaleImagePath, outputImagePath, outputImagePathGPU);

    unsigned char *c_image = stbi_load(coloredImagePath, &c_width, &c_height, &c_bpp, imgchannels );
    unsigned char *c_as_g_image = stbi_load(coloredAsGrayscaleImagePath, &c_as_g_width, &c_as_g_height, &c_as_g_bpp, 1 );
    unsigned char *g_image = stbi_load(grayscaleImagePath, &g_width, &g_height, &g_bpp, 1 );
    finalImage = (unsigned char*) malloc(3 * g_width * g_height * sizeof(unsigned char));
    finalImageByGPU = (unsigned char*) malloc(3 * g_width * g_height * sizeof(unsigned char));
    memset(finalImage, '\0', 3 * g_width * g_height * sizeof(unsigned char));
    memset(finalImageByGPU, '\0', 3 * g_width * g_height * sizeof(unsigned char));
    copyGrayscaleToFinal(finalImage, g_image, g_width, g_height);
    copyGrayscaleToFinal(finalImageByGPU, g_image, g_width, g_height);

    // Convert colored image to grayscale
    // convertToGrayscale(c_as_g_image_load, c_image, c_width, c_height);
    // stbi_write_jpg(coloredAsGrayscaleImagePath, c_height, c_width, 1, c_as_g_image_load, 0);
    // unsigned char *c_as_g_image = stbi_load(coloredAsGrayscaleImagePath, &c_as_g_width, &c_as_g_height, &c_as_g_bpp, 1 );
    // printf("asdf: %d\n", (int)c_as_g_image[0]);
    // printf("asdf\n");

    // clock_t t = clock();
    // patchMatch(c_image, c_as_g_image, g_image, finalImage, maskCols, maskRows, c_width, c_height);
    // patchMatchEachPixel(c_image, c_as_g_image, g_image, finalImage, maskCols, maskRows, c_width, c_height);

    // t = clock() - t;

    // double time_taken = ((double)t) / CLOCKS_PER_SEC;
    // printf("Time Taken: %fms\n", time_taken * 1000);


    // GPU CODE HERE
    // Mallocs
    hipMalloc(&d_c_image, 3 * c_width * c_height * sizeof(unsigned char));
    hipMalloc(&d_c_as_g_image, c_width * c_height * sizeof(unsigned char));
    hipMalloc(&d_g_image, c_width * c_height * sizeof(unsigned char));
    hipMalloc(&d_finalImage, 3 * c_width * c_height * sizeof(unsigned char));
    hipMemset(d_finalImage, '\0', 3 * g_width * g_height * sizeof(unsigned char));

    // Memcpy
    hipMemcpy(d_c_image, c_image, 3 * c_width * c_height * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_c_as_g_image, c_as_g_image, c_width * c_height * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_g_image, g_image, c_width * c_height * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_finalImage, finalImageByGPU, 3 *  c_width * c_height * sizeof(unsigned char), hipMemcpyHostToDevice);



    int numOfThreadSq = BLOCKSIZESQ;
    dim3 threadsPerBlock(numOfThreadSq, numOfThreadSq);
    dim3 numOfBlocks(g_width/numOfThreadSq, g_height/numOfThreadSq);

    hipEventCreate(&start);
    hipEventRecord(start,0);

    gpuPathMatchEachPixel<<<numOfBlocks, threadsPerBlock>>>(d_c_image, d_c_as_g_image, d_g_image, d_finalImage, maskCols, maskRows, c_width, c_height, g_width, g_height);


    hipEventCreate(&stop);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("Elapsed time : %fms\n" ,elapsedTime);
    
    hipMemcpy(finalImageByGPU, d_finalImage, 3 *  c_width * c_height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    stbi_write_jpg(outputImagePath, g_height, g_width, 3, finalImage, 0);
    stbi_write_jpg(outputImagePathGPU, g_height, g_width, 3, finalImageByGPU, 0);

    free(finalImage);
    hipFree(d_c_image);
    hipFree(d_c_as_g_image);
    hipFree(d_g_image);
    hipFree(d_finalImage);
    return 0;
}
